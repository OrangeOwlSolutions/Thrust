#include <thrust/device_vector.h>
#include <thrust/reduce.h>
#include <thrust/random.h>
#include <thrust/sort.h>
#include <thrust/unique.h>
#include <thrust/equal.h>

#include <hipblas.h>

#include "Utilities.cuh"
#include "TimingGPU.cuh"

/**************************************************************/
/* CONVERT LINEAR INDEX TO ROW INDEX - NEEDED FOR APPROACH #1 */
/**************************************************************/
template <typename T>
struct linear_index_to_row_index : public thrust::unary_function<T,T> {
	
	T Ncols; // --- Number of columns
  
	__host__ __device__ linear_index_to_row_index(T Ncols) : Ncols(Ncols) {}

	__host__ __device__ T operator()(T i) { return i / Ncols; }
};

/***********************/
/* RECIPROCAL OPERATOR */
/***********************/
struct Inv: public thrust::unary_function<float, float>
{
    __host__ __device__ float operator()(float x)
    {
        return 1.0f / x;
    }
};

/********/
/* MAIN */
/********/
int main()
{
	/**************************/
	/* SETTING UP THE PROBLEM */
	/**************************/
  
	const int Nrows = 10;			// --- Number of rows
	const int Ncols =  3;			// --- Number of columns  

	// --- Random uniform integer distribution between 0 and 100
	thrust::default_random_engine rng;
	thrust::uniform_int_distribution<int> dist1(0, 100);

	// --- Random uniform integer distribution between 1 and 4
	thrust::uniform_int_distribution<int> dist2(1, 4);

	// --- Matrix allocation and initialization
	thrust::device_vector<float> d_matrix(Nrows * Ncols);
	for (size_t i = 0; i < d_matrix.size(); i++) d_matrix[i] = (float)dist1(rng);

	// --- Normalization vector allocation and initialization
	thrust::device_vector<float> d_normalization(Nrows);
	for (size_t i = 0; i < d_normalization.size(); i++) d_normalization[i] = (float)dist2(rng);

	printf("\n\nOriginal matrix\n");
	for(int i = 0; i < Nrows; i++) {
		std::cout << "[ ";
		for(int j = 0; j < Ncols; j++)
			std::cout << d_matrix[i * Ncols + j] << " ";
		std::cout << "]\n";
	}

	printf("\n\nNormlization vector\n");
	for(int i = 0; i < Nrows; i++) std::cout << d_normalization[i] << "\n";

	TimingGPU timerGPU;
	
	/*********************************/
	/* ROW NORMALIZATION WITH THRUST */
	/*********************************/

	thrust::device_vector<float> d_matrix2(d_matrix);

	timerGPU.StartCounter();
	thrust::transform(d_matrix2.begin(), d_matrix2.end(),
					  thrust::make_permutation_iterator(
								d_normalization.begin(),
								thrust::make_transform_iterator(thrust::make_counting_iterator(0), linear_index_to_row_index<int>(Ncols))),
					  d_matrix2.begin(),
					  thrust::divides<float>());
	std::cout << "Timing - Thrust = " << timerGPU.GetCounter() << "\n";
	
	printf("\n\nNormalized matrix - Thrust case\n");
	for(int i = 0; i < Nrows; i++) {
		std::cout << "[ ";
		for(int j = 0; j < Ncols; j++)
			std::cout << d_matrix2[i * Ncols + j] << " ";
		std::cout << "]\n";
	}

	/*********************************/
	/* ROW NORMALIZATION WITH CUBLAS */
	/*********************************/
	d_matrix2 = d_matrix;

	hipblasHandle_t handle;
	cublasSafeCall(hipblasCreate(&handle));

	timerGPU.StartCounter();
	thrust::transform(d_normalization.begin(), d_normalization.end(), d_normalization.begin(), Inv());
	cublasSafeCall(hipblasSdgmm(handle, HIPBLAS_SIDE_RIGHT, Ncols, Nrows, thrust::raw_pointer_cast(&d_matrix2[0]), Ncols, 
		           thrust::raw_pointer_cast(&d_normalization[0]), 1, thrust::raw_pointer_cast(&d_matrix2[0]), Ncols));
	std::cout << "Timing - cuBLAS = " << timerGPU.GetCounter() << "\n";

	printf("\n\nNormalized matrix - cuBLAS case\n");
	for(int i = 0; i < Nrows; i++) {
		std::cout << "[ ";
		for(int j = 0; j < Ncols; j++)
			std::cout << d_matrix2[i * Ncols + j] << " ";
		std::cout << "]\n";
	}

	return 0;
}
