#include <hipblas.h>

#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <thrust/generate.h>
#include <thrust/reduce.h>
#include <thrust/functional.h>
#include <thrust/random.h>
#include <thrust/sequence.h>

#include <stdio.h>
#include <iostream>

#include "Utilities.cuh"
#include "TimingGPU.cuh"

// --- Required for approach #2
__device__ float *vals;

/**************************************************************/
/* CONVERT LINEAR INDEX TO ROW INDEX - NEEDED FOR APPROACH #1 */
/**************************************************************/
template <typename T>
struct linear_index_to_row_index : public thrust::unary_function<T,T> {
	
	T Ncols; // --- Number of columns
  
	__host__ __device__ linear_index_to_row_index(T Ncols) : Ncols(Ncols) {}

	__host__ __device__ T operator()(T i) { return i / Ncols; }
};

/******************************************/
/* ROW_REDUCTION - NEEDED FOR APPROACH #2 */
/******************************************/
struct row_reduction {
  
	const int Ncols;	// --- Number of columns

	row_reduction(int _Ncols) : Ncols(_Ncols) {}

	__device__ float operator()(float& x, int& y ) {
		float temp = 0.f;
		for (int i = 0; i<Ncols; i++)
			temp += vals[i + (y*Ncols)];
		return temp;
	}
};

/********/
/* MAIN */
/********/
int main()
{
	const int Nrows = 5;     // --- Number of rows
	const int Ncols = 8;     // --- Number of columns
  
	// --- Random uniform integer distribution between 10 and 99
	thrust::default_random_engine rng;
	thrust::uniform_int_distribution<int> dist(10, 99);

	// --- Matrix allocation and initialization
	thrust::device_vector<float> d_matrix(Nrows * Ncols);
	for (size_t i = 0; i < d_matrix.size(); i++) d_matrix[i] = (float)dist(rng);
  
	TimingGPU timerGPU;

	/***************/
	/* APPROACH #1 */
	/***************/
	timerGPU.StartCounter();
	// --- Allocate space for row sums and indices
	thrust::device_vector<float> d_row_sums(Nrows);
	thrust::device_vector<int> d_row_indices(Nrows);
  
	// --- Compute row sums by summing values with equal row indices
	thrust::reduce_by_key(thrust::make_transform_iterator(thrust::counting_iterator<int>(0), linear_index_to_row_index<int>(Ncols)),
						  thrust::make_transform_iterator(thrust::counting_iterator<int>(0), linear_index_to_row_index<int>(Ncols)) + (Nrows*Ncols),
						  d_matrix.begin(),
						  d_row_indices.begin(),
						  d_row_sums.begin(),
						  thrust::equal_to<int>(),
						  thrust::plus<float>());

	printf("Timing for approach #1 = %f\n", timerGPU.GetCounter());
	
	// --- Print result
	for(int i = 0; i < Nrows; i++) {
		std::cout << "[ ";
		for(int j = 0; j < Ncols; j++)
			std::cout << d_matrix[i * Ncols + j] << " ";
		std::cout << "] = " << d_row_sums[i] << "\n";
	}

	/***************/
	/* APPROACH #2 */
	/***************/
	timerGPU.StartCounter();
	thrust::device_vector<float> d_row_sums_2(Nrows, 0);
	float *s_vals = thrust::raw_pointer_cast(&d_matrix[0]);
	gpuErrchk(hipMemcpyToSymbol(HIP_SYMBOL(vals), &s_vals, sizeof(float *)));
	thrust::transform(d_row_sums_2.begin(), d_row_sums_2.end(), thrust::counting_iterator<int>(0),  d_row_sums_2.begin(), row_reduction(Ncols));

	printf("Timing for approach #2 = %f\n", timerGPU.GetCounter());

	for(int i = 0; i < Nrows; i++) {
		std::cout << "[ ";
		for(int j = 0; j < Ncols; j++)
			std::cout << d_matrix[i * Ncols + j] << " ";
		std::cout << "] = " << d_row_sums_2[i] << "\n";
	}

	/***************/
	/* APPROACH #3 */
	/***************/
	hipblasHandle_t handle;

	timerGPU.StartCounter();
	cublasSafeCall(hipblasCreate(&handle));

	thrust::device_vector<float> d_row_sums_3(Nrows);
	thrust::device_vector<float> d_ones(Ncols, 1.f);

	float alpha = 1.f;
	float beta  = 0.f;
	cublasSafeCall(hipblasSgemv(handle, HIPBLAS_OP_T, Ncols, Nrows, &alpha, thrust::raw_pointer_cast(d_matrix.data()), Ncols, 
		                       thrust::raw_pointer_cast(d_ones.data()), 1, &beta, thrust::raw_pointer_cast(d_row_sums_3.data()), 1));

	printf("Timing for approach #3 = %f\n", timerGPU.GetCounter());

	for(int i = 0; i < Nrows; i++) {
		std::cout << "[ ";
		for(int j = 0; j < Ncols; j++)
			std::cout << d_matrix[i * Ncols + j] << " ";
		std::cout << "] = " << d_row_sums_3[i] << "\n";
	}

	return 0;
}

