#include <hipblas.h>

#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <thrust/generate.h>
#include <thrust/reduce.h>
#include <thrust/functional.h>
#include <thrust/random.h>
#include <thrust/sequence.h>

#include <stdio.h>
#include <iostream>

#include "Utilities.cuh"
#include "TimingGPU.cuh"

// --- Required for approach #2
__device__ float *vals;

/**************************************************************/
/* CONVERT LINEAR INDEX TO ROW INDEX - NEEDED FOR APPROACH #1 */
/**************************************************************/
template <typename T>
struct linear_index_to_row_index : public thrust::unary_function<T,T> {
	
	T Ncols; // --- Number of columns
  
	__host__ __device__ linear_index_to_row_index(T Ncols) : Ncols(Ncols) {}

	__host__ __device__ T operator()(T i) { return i / Ncols; }
};

/******************************************/
/* ROW_REDUCTION - NEEDED FOR APPROACH #2 */
/******************************************/
struct row_reduction {
  
	const int Ncols;	// --- Number of columns

	row_reduction(int _Ncols) : Ncols(_Ncols) {}

	__device__ float operator()(float& x, int& y ) {
		float temp = 0.f;
		for (int i = 0; i<Ncols; i++)
			temp += vals[i + (y*Ncols)];
		return temp;
	}
};

/**************************/
/* NEEDED FOR APPROACH #3 */
/**************************/
template<typename T>
struct MulC: public thrust::unary_function<T, T>
{
    T C;
    __host__ __device__ MulC(T c) : C(c) { }
    __host__ __device__ T operator()(T x) { return x * C; }
};

/********/
/* MAIN */
/********/
int main()
{
	const int Nrows = 5;     // --- Number of rows
	const int Ncols = 8;     // --- Number of columns
  
	// --- Random uniform integer distribution between 10 and 99
	thrust::default_random_engine rng;
	thrust::uniform_int_distribution<int> dist(10, 99);

	// --- Matrix allocation and initialization
	thrust::device_vector<float> d_matrix(Nrows * Ncols);
	for (size_t i = 0; i < d_matrix.size(); i++) d_matrix[i] = (float)dist(rng);
  
	TimingGPU timerGPU;

	/***************/
	/* APPROACH #1 */
	/***************/
	timerGPU.StartCounter();
	// --- Allocate space for row sums and indices
	thrust::device_vector<float> d_row_sums(Nrows);
	thrust::device_vector<int> d_row_indices(Nrows);
  
	// --- Compute row sums by summing values with equal row indices
	//thrust::reduce_by_key(thrust::make_transform_iterator(thrust::counting_iterator<int>(0), linear_index_to_row_index<int>(Ncols)),
	//					  thrust::make_transform_iterator(thrust::counting_iterator<int>(0), linear_index_to_row_index<int>(Ncols)) + (Nrows*Ncols),
	//					  d_matrix.begin(),
	//					  d_row_indices.begin(),
	//					  d_row_sums.begin(),
	//					  thrust::equal_to<int>(),
	//					  thrust::plus<float>());

	thrust::reduce_by_key(
                thrust::make_transform_iterator(thrust::make_counting_iterator(0), linear_index_to_row_index<int>(Ncols)),
                thrust::make_transform_iterator(thrust::make_counting_iterator(0), linear_index_to_row_index<int>(Ncols)) + (Nrows*Ncols),
                d_matrix.begin(),
                thrust::make_discard_iterator(),
                d_row_sums.begin());
	
	printf("Timing for approach #1 = %f\n", timerGPU.GetCounter());
	
	// --- Print result
	for(int i = 0; i < Nrows; i++) {
		std::cout << "[ ";
		for(int j = 0; j < Ncols; j++)
			std::cout << d_matrix[i * Ncols + j] << " ";
		std::cout << "] = " << d_row_sums[i] << "\n";
	}

	/***************/
	/* APPROACH #2 */
	/***************/
	timerGPU.StartCounter();
	thrust::device_vector<float> d_row_sums_2(Nrows, 0);
	float *s_vals = thrust::raw_pointer_cast(&d_matrix[0]);
	gpuErrchk(hipMemcpyToSymbol(HIP_SYMBOL(vals), &s_vals, sizeof(float *)));
	thrust::transform(d_row_sums_2.begin(), d_row_sums_2.end(), thrust::counting_iterator<int>(0),  d_row_sums_2.begin(), row_reduction(Ncols));

	printf("Timing for approach #2 = %f\n", timerGPU.GetCounter());

	for(int i = 0; i < Nrows; i++) {
		std::cout << "[ ";
		for(int j = 0; j < Ncols; j++)
			std::cout << d_matrix[i * Ncols + j] << " ";
		std::cout << "] = " << d_row_sums_2[i] << "\n";
	}

	/***************/
	/* APPROACH #3 */
	/***************/

	timerGPU.StartCounter();
	thrust::device_vector<float> d_row_sums_3(Nrows, 0);
	thrust::device_vector<float> d_temp(Nrows * Ncols);
	thrust::inclusive_scan_by_key(
                thrust::make_transform_iterator(thrust::make_counting_iterator(0), linear_index_to_row_index<int>(Ncols)),
                thrust::make_transform_iterator(thrust::make_counting_iterator(0), linear_index_to_row_index<int>(Ncols)) + (Nrows*Ncols),
                d_matrix.begin(),
                d_temp.begin());
    thrust::copy(
                thrust::make_permutation_iterator(
                        d_temp.begin() + Ncols - 1,
                        thrust::make_transform_iterator(thrust::make_counting_iterator(0), MulC<int>(Ncols))),
    thrust::make_permutation_iterator(
                        d_temp.begin() + Ncols - 1,
                        thrust::make_transform_iterator(thrust::make_counting_iterator(0), MulC<int>(Ncols))) + Nrows,
                d_row_sums_3.begin());
		
	printf("Timing for approach #3 = %f\n", timerGPU.GetCounter());

	for(int i = 0; i < Nrows; i++) {
		std::cout << "[ ";
		for(int j = 0; j < Ncols; j++)
			std::cout << d_matrix[i * Ncols + j] << " ";
		std::cout << "] = " << d_row_sums_3[i] << "\n";
	}

	/***************/
	/* APPROACH #4 */
	/***************/
	hipblasHandle_t handle;

	timerGPU.StartCounter();
	cublasSafeCall(hipblasCreate(&handle));

	thrust::device_vector<float> d_row_sums_4(Nrows);
	thrust::device_vector<float> d_ones(Ncols, 1.f);

	float alpha = 1.f;
	float beta  = 0.f;
	cublasSafeCall(hipblasSgemv(handle, HIPBLAS_OP_T, Ncols, Nrows, &alpha, thrust::raw_pointer_cast(d_matrix.data()), Ncols, 
		                       thrust::raw_pointer_cast(d_ones.data()), 1, &beta, thrust::raw_pointer_cast(d_row_sums_4.data()), 1));

	printf("Timing for approach #4 = %f\n", timerGPU.GetCounter());

	for(int i = 0; i < Nrows; i++) {
		std::cout << "[ ";
		for(int j = 0; j < Ncols; j++)
			std::cout << d_matrix[i * Ncols + j] << " ";
		std::cout << "] = " << d_row_sums_4[i] << "\n";
	}

	return 0;
}
