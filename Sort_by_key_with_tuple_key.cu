#include <thrust/device_vector.h>
#include <thrust/sort.h>

#include "Utilities.cuh"

// --- Defining tuple type
typedef thrust::tuple<int, int> Tuple;

/**************************/
/* TUPLE ORDERING FUNCTOR */
/**************************/
struct TupleComp
{
	__host__ __device__ bool operator()(const Tuple& t1, const Tuple& t2)
	{
		if (t1.get<0>() < t2.get<0>())
			return true;
		if (t1.get<0>() > t2.get<0>())
			return false;
		return t1.get<1>() < t2.get<1>();
	}
};

/********/
/* MAIN */
/********/
int main()
{
	const int N = 8;

	// --- Keys and values on the host: allocation and definition
	int h_keys1[N]		= { 1, 3, 3, 3, 2, 3, 2, 1 };                                         
	int h_keys2[N]		= { 1, 5, 3, 8, 2, 8, 1, 1 };                                         
	float h_values[N]	= { 0.3, 5.1, 3.2, -0.08, 2.1, 5.2, 1.1, 0.01};

	printf("\n\n");
	printf("Original\n");
	for (int i = 0; i < N; i++) {
		printf("%i %i %f\n", h_keys1[i], h_keys2[i], h_values[i]);
	}

	// --- Keys and values on the device: allocation
	int *d_keys1;		gpuErrchk(hipMalloc(&d_keys1, N * sizeof(int)));
	int *d_keys2;		gpuErrchk(hipMalloc(&d_keys2, N * sizeof(int)));
	float *d_values;	gpuErrchk(hipMalloc(&d_values, N * sizeof(float)));

	// --- Keys and values: host -> device
	gpuErrchk(hipMemcpy(d_keys1, h_keys1, N * sizeof(int), hipMemcpyHostToDevice));
	gpuErrchk(hipMemcpy(d_keys2, h_keys2, N * sizeof(int), hipMemcpyHostToDevice));
	gpuErrchk(hipMemcpy(d_values, h_values, N * sizeof(float), hipMemcpyHostToDevice));

	// --- From raw pointers to device_ptr
	thrust::device_ptr<int> dev_ptr_keys1 = thrust::device_pointer_cast(d_keys1);
	thrust::device_ptr<int> dev_ptr_keys2 = thrust::device_pointer_cast(d_keys2);
	thrust::device_ptr<float> dev_ptr_values = thrust::device_pointer_cast(d_values);

	// --- Declare outputs
	thrust::device_vector<float> d_values_output(N);
	thrust::device_vector<Tuple> d_keys_output(N);

	auto begin_keys = thrust::make_zip_iterator(thrust::make_tuple(dev_ptr_keys1, dev_ptr_keys2));
	auto end_keys = thrust::make_zip_iterator(thrust::make_tuple(dev_ptr_keys1 + N, dev_ptr_keys2 + N));

	thrust::sort_by_key(begin_keys, end_keys, dev_ptr_values, TupleComp());

	int *h_keys1_output = (int *)malloc(N * sizeof(int));
	int *h_keys2_output = (int *)malloc(N * sizeof(int));
	float *h_values_output = (float *)malloc(N * sizeof(float));

	gpuErrchk(hipMemcpy(h_keys1_output, d_keys1, N * sizeof(int), hipMemcpyDeviceToHost));
	gpuErrchk(hipMemcpy(h_keys2_output, d_keys2, N * sizeof(int), hipMemcpyDeviceToHost));
	gpuErrchk(hipMemcpy(h_values_output, d_values, N * sizeof(float), hipMemcpyDeviceToHost));

	printf("\n\n");
	printf("Ordered\n");
	for (int i = 0; i < N; i++) {
		printf("%i %i %f\n", h_keys1_output[i], h_keys2_output[i], h_values_output[i]);
	}

}
