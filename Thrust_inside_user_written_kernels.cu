#include "hip/hip_runtime.h"
#include <stdio.h>

#include <thrust/reduce.h>
#include <thrust/execution_policy.h>

#include "TimingGPU.cuh"
#include "Utilities.cuh"

#define BLOCKSIZE_1D	256
#define BLOCKSIZE_2D_X	32
#define BLOCKSIZE_2D_Y	32

/*************************/
/* TEST KERNEL FUNCTIONS */
/*************************/
__global__ void test1(const float * __restrict__ d_data, float * __restrict__ d_results, const int Nrows, const int Ncols) {

	const unsigned int tid = threadIdx.x + blockDim.x * blockIdx.x;

	if (tid < Nrows) d_results[tid] = thrust::reduce(thrust::seq, d_data + tid * Ncols, d_data + (tid + 1) * Ncols);

}

__global__ void test2(const float * __restrict__ d_data, float * __restrict__ d_results, const int Nrows, const int Ncols) {

	const unsigned int tid = threadIdx.x + blockDim.x * blockIdx.x;

	if (tid < Nrows) d_results[tid] = thrust::reduce(thrust::device, d_data + tid * Ncols, d_data + (tid + 1) * Ncols);

}

/********/
/* MAIN */
/********/
int main() {

    const int Nrows = 64;
    const int Ncols = 2048;

    gpuErrchk(hipFree(0));

//    size_t DevQueue;
//    gpuErrchk(hipDeviceGetLimit(&DevQueue, cudaLimitDevRuntimePendingLaunchCount));
//    DevQueue *= 128;
//    gpuErrchk(hipDeviceSetLimit(cudaLimitDevRuntimePendingLaunchCount, DevQueue));

    float *h_data		= (float *)malloc(Nrows * Ncols * sizeof(float));
	float *h_results	= (float *)malloc(Nrows *         sizeof(float));
	float *h_results1	= (float *)malloc(Nrows *         sizeof(float));
	float *h_results2	= (float *)malloc(Nrows *         sizeof(float));
	float sum = 0.f;
    for (int i=0; i<Nrows; i++) {
		h_results[i] = 0.f;
	    for (int j=0; j<Ncols; j++) {
			h_data[i*Ncols+j] = i;
			h_results[i] = h_results[i] + h_data[i*Ncols+j];
		}
	}

	TimingGPU timerGPU;

	float *d_data;			gpuErrchk(hipMalloc((void**)&d_data,	  Nrows * Ncols * sizeof(float)));
    float *d_results1;		gpuErrchk(hipMalloc((void**)&d_results1, Nrows         * sizeof(float)));
    float *d_results2;		gpuErrchk(hipMalloc((void**)&d_results2, Nrows         * sizeof(float)));
    gpuErrchk(hipMemcpy(d_data, h_data, Nrows * Ncols * sizeof(float), hipMemcpyHostToDevice));

	timerGPU.StartCounter();
	test1<<<iDivUp(Nrows, BLOCKSIZE_1D), BLOCKSIZE_1D>>>(d_data, d_results1, Nrows, Ncols);
	gpuErrchk(hipPeekAtLastError());
	gpuErrchk(hipDeviceSynchronize());
	printf("Timing approach nr. 1 = %f\n", timerGPU.GetCounter());

	gpuErrchk(hipMemcpy(h_results1, d_results1, Nrows * sizeof(float), hipMemcpyDeviceToHost));

	for (int i=0; i<Nrows; i++) {
		if (h_results1[i] != h_results[i]) {
			printf("Approach nr. 1; Error at i = %i; h_results1 = %f; h_results = %f", i, h_results1[i], h_results[i]);
			return 0;
		}
	}

	timerGPU.StartCounter();
	test2<<<iDivUp(Nrows, BLOCKSIZE_1D), BLOCKSIZE_1D>>>(d_data, d_results1, Nrows, Ncols);
	gpuErrchk(hipPeekAtLastError());
	gpuErrchk(hipDeviceSynchronize());
	printf("Timing approach nr. 2 = %f\n", timerGPU.GetCounter());

	gpuErrchk(hipMemcpy(h_results1, d_results1, Nrows * sizeof(float), hipMemcpyDeviceToHost));

	for (int i=0; i<Nrows; i++) {
		if (h_results1[i] != h_results[i]) {
			printf("Approach nr. 2; Error at i = %i; h_results1 = %f; h_results = %f", i, h_results1[i], h_results[i]);
			return 0;
		}
	}

    printf("Test passed!\n");

}
