#include "hip/hip_runtime.h"
#include <thrust\device_vector.h>
#include <thrust\transform_reduce.h>
#include <thrust\sequence.h>
#include <thrust\random.h>
#include <thrust\gather.h>
#include <thrust\extrema.h>

using namespace thrust::placeholders;

/****************************************************/
/* POWER DIFFERENCE FUNCTOR FOR EUCLIDEAN DISTANCES */
/****************************************************/
struct PowerDifference {
	__host__ __device__ float operator()(const float& a, const float& b) const { return pow(a - b, 2); }
};

/*******************/
/* EXPAND OPERATOR */
/*******************/
template <typename InputIterator1, typename InputIterator2, typename OutputIterator>
OutputIterator expand(InputIterator1 first1,
                      InputIterator1 last1,
                      InputIterator2 first2,
                      OutputIterator output)
{
	typedef typename thrust::iterator_difference<InputIterator1>::type difference_type;
  
	difference_type input_size  = thrust::distance(first1, last1);
	difference_type output_size = thrust::reduce(first1, last1);

	// scan the counts to obtain output offsets for each input element
	thrust::device_vector<difference_type> output_offsets(input_size, 0);
	thrust::exclusive_scan(first1, last1, output_offsets.begin()); 

	// scatter the nonzero counts into their corresponding output positions
	thrust::device_vector<difference_type> output_indices(output_size, 0);
	thrust::scatter_if(thrust::counting_iterator<difference_type>(0), thrust::counting_iterator<difference_type>(input_size),
					   output_offsets.begin(), first1, output_indices.begin());

	// compute max-scan over the output indices, filling in the holes
	thrust::inclusive_scan(output_indices.begin(), output_indices.end(), output_indices.begin(), thrust::maximum<difference_type>());

	// gather input values according to index array (output = first2[output_indices])
	OutputIterator output_end = output; thrust::advance(output_end, output_size);
	thrust::gather(output_indices.begin(), output_indices.end(), first2, output);

	// return output + output_size
	thrust::advance(output, output_size);
  
	return output;
}

/********/
/* MAIN */
/********/
int main()
{
	/**************************/
	/* SETTING UP THE PROBLEM */
	/**************************/
  
	const int N		= 10;			// --- Number of vector elements
	const int Nvec	= 20;			// --- Number of vectors

	// --- Random uniform integer distribution between 0 and 100
	thrust::default_random_engine rng;
	thrust::uniform_int_distribution<int> dist(0, 20);

	// --- Matrix allocation and initialization
	thrust::device_vector<float> d_vec(N);
	thrust::device_vector<float> d_test_vectors(Nvec * N);
	for (size_t i = 0; i < d_vec.size(); i++) d_vec[i] = (float)dist(rng);
	for (size_t i = 0; i < d_test_vectors.size(); i++) d_test_vectors[i] = (float)dist(rng);

	printf("\n\nVectors\n");
	for(int i = 0; i < N; i++) {
			std::cout << d_vec[i] << " ";
	}

	printf("\n\nTest vectors\n");
	for(int i = 0; i < Nvec; i++) {
		std::cout << " [ ";
		for(int j = 0; j < N; j++)
			std::cout << d_test_vectors[i * N + j] << " ";
		std::cout << "]\n";
	}

	/*******************************************************************************/
	/* CALCULATING THE EUCLIDEAN DISTANCES BETWEEN THE VECTOR AND THE TEST VECTORS */
	/*******************************************************************************/
	// --- Creating the indices for the reduction by key
	thrust::device_vector<int> d_sequence(Nvec);
	thrust::device_vector<int> d_indices(Nvec * N);
	thrust::device_vector<int> d_counts(Nvec, N);
	thrust::sequence(d_sequence.begin(), d_sequence.begin() + Nvec);
	expand(d_counts.begin(), d_counts.end(), d_sequence.begin(), d_indices.begin());

	thrust::device_vector<int> d_counts2(N, Nvec);
	thrust::device_vector<float> d_squared_differences(Nvec * N);
    expand(d_counts2.begin(), d_counts2.end(), d_vec.begin(), 
        thrust::make_permutation_iterator(
                                d_squared_differences.begin(),
                                thrust::make_transform_iterator(thrust::make_counting_iterator(0), (_1 % Nvec) * N + _1 / Nvec)));

	thrust::transform(d_squared_differences.begin(), d_squared_differences.end(), d_test_vectors.begin(), d_squared_differences.begin(), PowerDifference());

	thrust::device_vector<float> d_norms(Nvec);
	thrust::reduce_by_key(d_indices.begin(), d_indices.end(), d_squared_differences.begin(), d_indices.begin(), d_norms.begin());
	
	thrust::device_vector<float>::iterator iter = thrust::min_element(d_norms.begin(), d_norms.end());
	unsigned int position = iter - d_norms.begin();

	printf("\n\ndnorms\n");
	for(int i = 0; i < Nvec; i++) {
			std::cout << d_norms[i] << " ";
	}

	printf("\n\nThe minimum distance element is: %i\n", position);

	return 0; 
}
