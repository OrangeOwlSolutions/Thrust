#include "hip/hip_runtime.h"
#include <iostream>

#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>

#include <thrust\device_vector.h>
#include <thrust\execution_policy.h>

#include "Utilities.cuh"

using namespace std;

#define NUM_THREADS 32
#define NUM_BLOCKS 16
#define NUM_STREAMS 3

struct BinaryOp{ __host__ __device__ int operator()(const int& o1,const int& o2) { return o1 * o2; } };

int main()
{
    const int N = 6000000;
    
    // --- Host side input data allocation and initialization. Registering host memory as page-locked (required for asynch hipMemcpyAsync).
    int *h_in = new int[N];	for(int i = 0; i < N; i++) h_in[i] = 5;
    gpuErrchk(hipHostRegister(h_in, N * sizeof(int), hipHostRegisterPortable));

    // --- Host side input data allocation and initialization. Registering host memory as page-locked (required for asynch hipMemcpyAsync).
    int *h_out = new int[N]; for(int i = 0; i < N; i++) h_out[i] = 0;
	gpuErrchk(hipHostRegister(h_out, N * sizeof(int), hipHostRegisterPortable));

    // --- Host side check results vector allocation and initialization
    int *h_checkResults = new int[N]; for(int i = 0; i < N; i++) h_checkResults[i] = h_in[i] * h_in[i];

    // --- Device side input data allocation.
    int *d_in = 0;				gpuErrchk(hipMalloc((void **)&d_in, N * sizeof(int)));

    // --- Device side output data allocation. 
	int *d_out = 0;				gpuErrchk( hipMalloc((void **)&d_out, N * sizeof(int)));
    
	int streamSize = N / NUM_STREAMS;
    size_t streamMemSize = N * sizeof(int) / NUM_STREAMS;

	// --- Set kernel launch configuration
    dim3 nThreads		= dim3(NUM_THREADS,1,1);
    dim3 nBlocks		= dim3(NUM_BLOCKS, 1,1);
    dim3 subKernelBlock = dim3((int)ceil((float)nBlocks.x / 2));

    // --- Create CUDA streams
    hipStream_t streams[NUM_STREAMS];
    for(int i = 0; i < NUM_STREAMS; i++)
        gpuErrchk(hipStreamCreate(&streams[i]));

	/**************************/
	/* BREADTH-FIRST APPROACH */
	/**************************/
	
	for(int i = 0; i < NUM_STREAMS; i++) {
        int offset = i * streamSize;
        hipMemcpyAsync(&d_in[offset], &h_in[offset], streamMemSize, hipMemcpyHostToDevice,     streams[i]);
	}

	for(int i = 0; i < NUM_STREAMS; i++)
    {
        int offset = i * streamSize;
        
		thrust::transform(thrust::cuda::par.on(streams[i]), thrust::device_pointer_cast(&d_in[offset]), thrust::device_pointer_cast(&d_in[offset]) + streamSize/2, 
			                                                thrust::device_pointer_cast(&d_in[offset]), thrust::device_pointer_cast(&d_out[offset]), BinaryOp());
		thrust::transform(thrust::cuda::par.on(streams[i]), thrust::device_pointer_cast(&d_in[offset + streamSize/2]), thrust::device_pointer_cast(&d_in[offset + streamSize/2]) + streamSize/2, 
			                                                thrust::device_pointer_cast(&d_in[offset + streamSize/2]), thrust::device_pointer_cast(&d_out[offset + streamSize/2]), BinaryOp());
		
    }

	for(int i = 0; i < NUM_STREAMS; i++) {
        int offset = i * streamSize;
        hipMemcpyAsync(&h_out[offset], &d_out[offset], streamMemSize, hipMemcpyDeviceToHost,   streams[i]);
	}

    for(int i = 0; i < NUM_STREAMS; i++)
        gpuErrchk(hipStreamSynchronize(streams[i]));

    gpuErrchk(hipDeviceSynchronize());

    // --- Release resources
    gpuErrchk(hipHostUnregister(h_in));
    gpuErrchk(hipHostUnregister(h_out));
    gpuErrchk(hipFree(d_in));
    gpuErrchk(hipFree(d_out));

    for(int i = 0; i < NUM_STREAMS; i++)
        gpuErrchk(hipStreamDestroy(streams[i]));

    hipDeviceReset();  

    // --- GPU output check
    int sum = 0;
    for(int i = 0; i < N; i++) {     
        //printf("%i %i\n", h_out[i], h_checkResults[i]);
		sum += h_checkResults[i] - h_out[i];
	}

    cout << "Error between CPU and GPU: " << sum << endl;

    delete[] h_in;
    delete[] h_out;
    delete[] h_checkResults;

    return 0;
}
