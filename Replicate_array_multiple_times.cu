#include "hip/hip_runtime.h"
#include <thrust/device_vector.h>
#include <thrust/gather.h>
#include <thrust/sequence.h>
#include <stdio.h>

using namespace thrust::placeholders;

/*************************************/
/* CONVERT LINEAR INDEX TO ROW INDEX */
/*************************************/
template <typename T>
struct linear_index_to_row_index : public thrust::unary_function<T,T> {
	
	T Ncols; // --- Number of columns
  
	__host__ __device__ linear_index_to_row_index(T Ncols) : Ncols(Ncols) {}

	__host__ __device__ T operator()(T i) { return i / Ncols; }
};

/*******************/
/* EXPAND OPERATOR */
/*******************/
template <typename InputIterator1, typename InputIterator2, typename OutputIterator>
OutputIterator expand(InputIterator1 first1,
                      InputIterator1 last1,
                      InputIterator2 first2,
                      OutputIterator output)
{
	typedef typename thrust::iterator_difference<InputIterator1>::type difference_type;
  
	difference_type input_size  = thrust::distance(first1, last1);
	difference_type output_size = thrust::reduce(first1, last1);

	// scan the counts to obtain output offsets for each input element
	thrust::device_vector<difference_type> output_offsets(input_size, 0);
	thrust::exclusive_scan(first1, last1, output_offsets.begin()); 

	// scatter the nonzero counts into their corresponding output positions
	thrust::device_vector<difference_type> output_indices(output_size, 0);
	thrust::scatter_if(thrust::counting_iterator<difference_type>(0), thrust::counting_iterator<difference_type>(input_size),
					   output_offsets.begin(), first1, output_indices.begin());

	// compute max-scan over the output indices, filling in the holes
	thrust::inclusive_scan(output_indices.begin(), output_indices.end(), output_indices.begin(), thrust::maximum<difference_type>());

	// gather input values according to index array (output = first2[output_indices])
	OutputIterator output_end = output; thrust::advance(output_end, output_size);
	thrust::gather(output_indices.begin(), output_indices.end(), first2, output);

	// return output + output_size
	thrust::advance(output, output_size);
  
	return output;
}

/**************************/
/* STRIDED RANGE OPERATOR */
/**************************/
template <typename Iterator>
class strided_range
{
    public:

    typedef typename thrust::iterator_difference<Iterator>::type difference_type;

    struct stride_functor : public thrust::unary_function<difference_type,difference_type>
    {
        difference_type stride;

        stride_functor(difference_type stride)
            : stride(stride) {}

        __host__ __device__
        difference_type operator()(const difference_type& i) const
        {
            return stride * i;
        }
    };

    typedef typename thrust::counting_iterator<difference_type>                   CountingIterator;
    typedef typename thrust::transform_iterator<stride_functor, CountingIterator> TransformIterator;
    typedef typename thrust::permutation_iterator<Iterator,TransformIterator>     PermutationIterator;

    // type of the strided_range iterator
    typedef PermutationIterator iterator;

    // construct strided_range for the range [first,last)
    strided_range(Iterator first, Iterator last, difference_type stride)
        : first(first), last(last), stride(stride) {}

    iterator begin(void) const
    {
        return PermutationIterator(first, TransformIterator(CountingIterator(0), stride_functor(stride)));
    }

    iterator end(void) const
    {
        return begin() + ((last - first) + (stride - 1)) / stride;
    }

    protected:
    Iterator first;
    Iterator last;
    difference_type stride;
};

/********/
/* MAIN */
/********/
int main(){

	/**************************/
	/* SETTING UP THE PROBLEM */
	/**************************/
  
	const int Nrows = 10;			// --- Number of objects
	const int Ncols	=  3;			// --- Number of centroids  

	thrust::device_vector<int> d_sequence(Nrows * Ncols);
	thrust::device_vector<int> d_counts(Ncols, Nrows);
	thrust::sequence(d_sequence.begin(), d_sequence.begin() + Ncols);
	expand(d_counts.begin(), d_counts.end(), d_sequence.begin(), 
		thrust::make_permutation_iterator(
								d_sequence.begin(),
								thrust::make_transform_iterator(thrust::make_counting_iterator(0),(_1 % Nrows) * Ncols + _1 / Nrows)));

	printf("\n\nCentroid indices\n");
	for(int i = 0; i < Nrows; i++) {
		std::cout << " [ ";
		for(int j = 0; j < Ncols; j++)
			std::cout << d_sequence[i * Ncols + j] << " ";
		std::cout << "]\n";
	}

	return 0;
}
