#include <hipblas.h>

#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <thrust/generate.h>
#include <thrust/sort.h>
#include <thrust/functional.h>
#include <thrust/random.h>
#include <thrust/sequence.h>

#include <stdio.h>
#include <iostream>

#include "Utilities.cuh"

/**************************************************************/
/* CONVERT LINEAR INDEX TO ROW INDEX - NEEDED FOR APPROACH #1 */
/**************************************************************/
template <typename T>
struct linear_index_to_row_index : public thrust::unary_function<T,T> {
	
	T Ncols; // --- Number of columns
  
	__host__ __device__ linear_index_to_row_index(T Ncols) : Ncols(Ncols) {}

	__host__ __device__ T operator()(T i) { return i / Ncols; }
};

/********/
/* MAIN */
/********/
int main()
{
	const int Nrows = 5;     // --- Number of rows
	const int Ncols = 8;     // --- Number of columns
  
	// --- Random uniform integer distribution between 10 and 99
	thrust::default_random_engine rng;
	thrust::uniform_int_distribution<int> dist(10, 99);

	// --- Matrix allocation and initialization
	thrust::device_vector<float> d_matrix(Nrows * Ncols);
	for (size_t i = 0; i < d_matrix.size(); i++) d_matrix[i] = (float)dist(rng);
  
	// --- Print result
	printf("Original matrix\n");
	for(int i = 0; i < Nrows; i++) {
		std::cout << "[ ";
		for(int j = 0; j < Ncols; j++)
			std::cout << d_matrix[i * Ncols + j] << " ";
		std::cout << "]\n";
	}

	/*************************/
	/* BACK-TO-BACK APPROACH */
	/*************************/
	thrust::device_vector<float> d_keys(Nrows * Ncols);

	// --- Generate row indices
	thrust::transform(thrust::make_counting_iterator(0),
				      thrust::make_counting_iterator(Nrows*Ncols),
					  thrust::make_constant_iterator(Ncols),
					  d_keys.begin(),
					  thrust::divides<int>());

	// --- Back-to-back approach
	thrust::stable_sort_by_key(d_matrix.begin(),
						       d_matrix.end(),
						       d_keys.begin(),
						       thrust::less<float>());

	thrust::stable_sort_by_key(d_keys.begin(),
						       d_keys.end(),
						       d_matrix.begin(),
						       thrust::less<int>());

	// --- Print result
	printf("\n\nSorted matrix\n");
	for(int i = 0; i < Nrows; i++) {
		std::cout << "[ ";
		for(int j = 0; j < Ncols; j++)
			std::cout << d_matrix[i * Ncols + j] << " ";
		std::cout << "]\n";
	}

	return 0;
}
