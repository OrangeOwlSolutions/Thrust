#include "hip/hip_runtime.h"
#include "device_launch_paraMeters.h"

#include <thrust\device_vector.h>
#include <thrust/extrema.h>

/***********************/
/* CUDA ERROR CHECKING */
/***********************/
#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
   if (code != hipSuccess) 
   {
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}

/********/
/* MAIN */
/********/
int main() {

	srand(time(NULL));
	
	const int N = 10;

	float *h_vec = (float *)malloc(N * sizeof(float));
	for (int i=0; i<N; i++) {
		h_vec[i] = rand() / (float)(RAND_MAX);
		printf("h_vec[%i] = %f\n", i, h_vec[i]);
	}
	
	float *d_vec; gpuErrchk(hipMalloc((void**)&d_vec, N * sizeof(float)));
	gpuErrchk(hipMemcpy(d_vec, h_vec, N * sizeof(float), hipMemcpyHostToDevice));

    thrust::device_ptr<float> dev_ptr = thrust::device_pointer_cast(d_vec);

	thrust::device_ptr<float> min_ptr = thrust::min_element(dev_ptr, dev_ptr + N);
	
	float min_value = min_ptr[0];
	printf("\nMininum value = %f\n", min_value);
	printf("Position = %i\n", &min_ptr[0] - &dev_ptr[0]);

}
