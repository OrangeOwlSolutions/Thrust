#include "hip/hip_runtime.h"
#include <thrust\device_vector.h>
#include <thrust\transform_reduce.h>
#include <thrust\sequence.h>
#include <thrust\random.h>
#include <thrust\gather.h>
#include <thrust\extrema.h>

using namespace thrust::placeholders;

/****************************************************/
/* POWER DIFFERENCE FUNCTOR FOR EUCLIDEAN DISTANCES */
/****************************************************/
struct PowerDifference {
	__host__ __device__ float operator()(const float& a, const float& b) const { return pow(a - b, 2); }
};

/*******************/
/* EXPAND OPERATOR */
/*******************/
template <typename InputIterator1, typename InputIterator2, typename OutputIterator>
OutputIterator expand(InputIterator1 first1,
                      InputIterator1 last1,
                      InputIterator2 first2,
                      OutputIterator output)
{
	typedef typename thrust::iterator_difference<InputIterator1>::type difference_type;
  
	difference_type input_size  = thrust::distance(first1, last1);
	difference_type output_size = thrust::reduce(first1, last1);

	// scan the counts to obtain output offsets for each input element
	thrust::device_vector<difference_type> output_offsets(input_size, 0);
	thrust::exclusive_scan(first1, last1, output_offsets.begin()); 

	// scatter the nonzero counts into their corresponding output positions
	thrust::device_vector<difference_type> output_indices(output_size, 0);
	thrust::scatter_if(thrust::counting_iterator<difference_type>(0), thrust::counting_iterator<difference_type>(input_size),
					   output_offsets.begin(), first1, output_indices.begin());

	// compute max-scan over the output indices, filling in the holes
	thrust::inclusive_scan(output_indices.begin(), output_indices.end(), output_indices.begin(), thrust::maximum<difference_type>());

	// gather input values according to index array (output = first2[output_indices])
	OutputIterator output_end = output; thrust::advance(output_end, output_size);
	thrust::gather(output_indices.begin(), output_indices.end(), first2, output);

	// return output + output_size
	thrust::advance(output, output_size);
  
	return output;
}

/********/
/* MAIN */
/********/
int main()
{
	/**************************/
	/* SETTING UP THE PROBLEM */
	/**************************/
  
	const int N		= 10;			// --- Number of vector elements
	const int Nvec	= 20;			// --- Number of vectors for each matrix

	// --- Random uniform integer distribution between 0 and 100
	thrust::default_random_engine rng;
	thrust::uniform_int_distribution<int> dist(0, 20);

	// --- Matrix allocation and initialization
	thrust::device_vector<float> d_matrix1(Nvec * N);
	thrust::device_vector<float> d_matrix2(Nvec * N);
	for (size_t i = 0; i < d_matrix1.size(); i++) d_matrix1[i] = (float)dist(rng);
	for (size_t i = 0; i < d_matrix2.size(); i++) d_matrix2[i] = (float)dist(rng);

	printf("\n\nFirst matrix\n");
	for(int i = 0; i < Nvec; i++) {
		std::cout << " [ ";
		for(int j = 0; j < N; j++)
			std::cout << d_matrix1[i * N + j] << " ";
		std::cout << "]\n";
	}

	printf("\n\nSecond matrix\n");
	for(int i = 0; i < Nvec; i++) {
		std::cout << " [ ";
		for(int j = 0; j < N; j++)
			std::cout << d_matrix2[i * N + j] << " ";
		std::cout << "]\n";
	}

	/****************************************************************************/
	/* CALCULATING THE EUCLIDEAN DISTANCES BETWEEN THE ROWS OF THE TWO MATRICES */
	/****************************************************************************/
	// --- Creating the indices for the reduction by key
	thrust::device_vector<int> d_sequence(Nvec);
	thrust::device_vector<int> d_indices(Nvec * N);
	thrust::device_vector<int> d_counts(Nvec, N);
	thrust::sequence(d_sequence.begin(), d_sequence.begin() + Nvec);
	expand(d_counts.begin(), d_counts.end(), d_sequence.begin(), d_indices.begin());

	printf("\n\nSecond matrix\n");
	for(int i = 0; i < Nvec; i++) {
		std::cout << " [ ";
		for(int j = 0; j < N; j++)
			std::cout << d_indices[i * N + j] << " ";
		std::cout << "]\n";
	}

	thrust::device_vector<float> d_squared_differences(Nvec * N);

	thrust::transform(d_matrix1.begin(), d_matrix1.end(), d_matrix2.begin(), d_squared_differences.begin(), PowerDifference());

	thrust::device_vector<float> d_norms(Nvec);
	thrust::reduce_by_key(d_indices.begin(), d_indices.end(), d_squared_differences.begin(), d_indices.begin(), d_norms.begin());
	
	printf("\n\ndnorms\n");
	for(int i = 0; i < Nvec; i++) {
			std::cout << d_norms[i] << " ";
	}

	return 0; 
}
